#include "hip/hip_runtime.h"
//
// A simple function that squares all the elements of an array
// through a call to a CUDA kernel
//

#include "cudasquare.cuh"

// Square function kernel
// It squares all the elements of an array
// Arguments:
// a: array that must be squared in the GPU
// b: squared int array (output)
// n: number of elements of the array
__global__
void square_kernel(int *a, int *b, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    int val=a[i];
    b[i] = val*val;
  }
}

// Square function
// Arguments:
// ha: array that must be squared in the CPU
// ('h' stands for 'host'.)
// n: number of elements of the array
// Return value: squared int array
int *square(int *ha, int n)
{
  // Create result int arrays on the CPU.
  int *hb = new int[n];

  // Create corresponding int arrays on the GPU.
  // ('d' stands for 'device'.)
  int *da, *db;
  hipMalloc(&da, n*sizeof(int));
  hipMalloc(&db, n*sizeof(int));

  // Copy input data from array on CPU to array on GPU.
  hipMemcpy(da, ha, n*sizeof(int), hipMemcpyHostToDevice);

  // Launch GPU code with n threads, one per
  // array element.
  square_kernel<<<(n+1023)/1024, 1024>>>(da, db, n);

  // Copy output array from GPU back to CPU.
  hipMemcpy(hb, db, n*sizeof(int), hipMemcpyDeviceToHost);

  // Free up the arrays on the GPU.
  hipFree(da);
  hipFree(db);

  //return as output the resulting array
  return hb;
}
